#include "hip/hip_runtime.h"
// CUDA runtime
#include "../include/core_headers.cuh"
//#include "../include/bh_helper.cuh"

const float   expA  = 0.24499f;
const float   expB = -0.8325f;  // -1.66490f dividing by two here so I don't have to take the square root of the spatial frequency
//onst float   expB =  -1.66490f; // dividing by two here so I don't have to take the square root of the spatial frequency
const float   expC = 2.81410f;
const float   kvScale = 1.0f; //FIXME for other voltages (0.8 for 200)

// createb_ctf
__global__ void ctf(hipfftReal* a, uint2 dims, uint2 o_dims, ctfParams b_ctf, float2 fourierVoxelSize, 
                    bool calc_centered) 
{

  // Other dimensions?   
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  if (x >= dims.x) { return ; }
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  if (y >= dims.y) { return ; }

  float tmp_coord;
  float radius_sq;
  long output_IDX;
  float phi;


  // TODO add centered calc
  output_IDX = y*dims.x + x;

  if (calc_centered)
  {
    y -= o_dims.y;
    if ( ! b_ctf.doHalfGrid ) { x -= o_dims.x ;}
  }
  else
  {
    if (y > o_dims.y) { y = y - dims.y ; }
    if ( ! b_ctf.doHalfGrid  && x > o_dims.x) {x = x - dims.x ;}
  }
 


  // TODO this seems safe, buy could there be probs for x or y == 0??
  tmp_coord = (float)y*fourierVoxelSize.y;
  radius_sq = (float)x*fourierVoxelSize.x;



  phi = atan2f(tmp_coord,radius_sq); 

  radius_sq = radius_sq*radius_sq + tmp_coord*tmp_coord;

    // should the sign on the amplitude contrast be flipped?
  a[output_IDX] = sinf(b_ctf.cs_term*powf(radius_sq,2) - b_ctf.df_term*radius_sq*(b_ctf.defocus1 + b_ctf.defocus2 * cosf(2.0f * (phi-b_ctf.astigmatism_angle))) - b_ctf.amplitudeContrast);
                  



  if (b_ctf.doSqCTF )
  {
    // Is this any better (or worse) than pow?
    a[output_IDX] *= a[output_IDX];
  }


    
}

// createb_ctf
__global__ void ctf(hipfftReal* a, uint2 dims, uint2 o_dims, ctfParams b_ctf, float2 fourierVoxelSize, 
                    bool calc_centered, float radial_weight, float total_exposure) 
{




  // Other dimensions?   
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  if (x >= dims.x) { return ; }
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  if (y >= dims.y) { return ; }

  float tmp_coord;
  float radius_sq;
  long output_IDX;
  float phi;


  // TODO add centered calc
  output_IDX = y*dims.x + x;

  if (calc_centered)
  {
    y -= o_dims.y;
    if ( ! b_ctf.doHalfGrid ) { x -= o_dims.x ;}
  }
  else
  {
    if (y > o_dims.y) { y = y - dims.y ; }
    if ( ! b_ctf.doHalfGrid  && x > o_dims.x) {x = x - dims.x ;}
  }
 


  // TODO this seems safe, buy could there be probs for x or y == 0??
  tmp_coord = (float)y*fourierVoxelSize.y;
  radius_sq = (float)x*fourierVoxelSize.x;

//  // modify occupancy by Radial weight
//  radial_weight *= (fabsf((float)x));

  phi = atan2f(tmp_coord,radius_sq); 

  radius_sq = radius_sq*radius_sq + tmp_coord*tmp_coord;
   
  a[output_IDX] = sinf(b_ctf.cs_term*powf(radius_sq,2) - b_ctf.df_term*radius_sq*(b_ctf.defocus1 + b_ctf.defocus2 * cosf(2.0f * (phi-b_ctf.astigmatism_angle))) - b_ctf.amplitudeContrast);
                  
                  
  // if you add the radial weighting you will need to fix this.


  if (b_ctf.doSqCTF )
  {
    // Is this any better (or worse) than pow?
    a[output_IDX] *= a[output_IDX];
  }

 a[output_IDX] *= radial_weight* expf( (-0.5f * total_exposure) / (kvScale *(expA * powf(radius_sq, expB) + expC)));
    
}

