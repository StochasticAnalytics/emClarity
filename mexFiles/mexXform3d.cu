#include "hip/hip_runtime.h"
#include "include/core_headers.cuh"



#define MAX_EPSILON_ERROR 5e-3f



////////////////////////////////////////////////////////////////////////////////
// Constants


// Texture reference for 2D float texture
//texture<float, 3, hipReadModeElementType> tex;
const float EXTRAPVAL = 0.0f;


////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void transformKernel_FWD(hipTextureObject_t thisTexObj,
                                    float* outputData,
                                    int3 dims,
                                    float3 rm_1,
                                    float3 rm_2,
                                    float3 rm_3,
                                    float3 shifts,
                                    float extrapVal,
                                    bool doFwdXform,
                                    float3 size_shift)
{


  // calculate normalized texture coordinates
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  if (x >= dims.x) { return ; }
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  if (y >= dims.y) { return ; }
  int z = blockIdx.z*blockDim.z + threadIdx.z;
  if (z >= dims.z) { return ; }

  float u,v,w,tu,tv,tw;

  if (doFwdXform)
  {

    // First, put the origin at the center, rotate, then shift
    u = (float)x - (float)(dims.x/2) + shifts.x + size_shift.x; 
    v = (float)y - (float)(dims.y/2) + shifts.y + size_shift.y; 
    w = (float)z - (float)(dims.z/2) + shifts.z + size_shift.z;
    tu = u*rm_1.x + v*rm_1.y + w*rm_1.z; 
    tv = u*rm_2.x + v*rm_2.y + w*rm_2.z;
    tw = u*rm_3.x + v*rm_3.y + w*rm_3.z;
  }
  else  
  {
    // First, put the origin at the center, shift, then rotate
    u = (float)x - (float)(dims.x/2); 
    v = (float)y - (float)(dims.y/2); 
    w = (float)z - (float)(dims.z/2);
    tu = u*rm_1.x + v*rm_1.y + w*rm_1.z + shifts.x + size_shift.x; 
    tv = u*rm_2.x + v*rm_2.y + w*rm_2.z + shifts.y + size_shift.y;
    tw = u*rm_3.x + v*rm_3.y + w*rm_3.z + shifts.z + size_shift.z;
  }

  // Convert to normalized coordinates 
  tu /= (float)dims.x; 
  tv /= (float)dims.y; 
  tw /= (float)dims.z;
  tu += 0.5f;
  tv += 0.5f;
  tw += 0.5f;

  if (tu < 0 | tv < 0 | tw < 0 | tu > 1 | tv > 1 | tw > 1)
  {

    outputData[ (z*dims.y + y) * dims.x + x ] = extrapVal;
  }
  else
  {
    outputData[ (z*dims.y + y) * dims.x + x ] = tex3D<float>(thisTexObj, tu, tv, tw);
  }

}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{

  float * d_output_img = NULL;
  float * d_input_img  = NULL;
  int3 dims;
  float3 shifts;
  float3 rm_1;
  float3 rm_2;
  float3 rm_3;
  bool *doFwdXform; // rotate then shift



  // We want to be able to re-use the texture object, so only set it up once.
  bool make_texture_obj = true;
  hipTextureObject_t* tex;

/* Check for proper number of arguments. TODO add checks on narg and types*/

  if ( ! mxGPUIsValidGPUData(prhs[0])) 
  {
        mexErrMsgIdAndTxt("MATLAB:mexFFT:rhs", "Expected a 3d gpuarray.");  
  }



    if ( nrhs > 4 )
    {

        mexPrintf("Re-using the tex object\n");
        tex    = (hipTextureObject_t *) mxGetData(prhs[4]);     

    }
        // This should be the input array or the the original arrays dims
      mxGPUArray const * inputArray  = mxGPUCreateFromMxArray(prhs[0]);
      mwSize const   input_dims = mxGPUGetNumberOfDimensions(inputArray);
      mwSize const * input_size = mxGPUGetDimensions(inputArray);
      mwSize const   numel_input = mxGPUGetNumberOfElements(inputArray);
      mxComplexity   input_data_type = mxGPUGetComplexity(inputArray);
    



  


  mxGPUArray * outputArray;

  // The first arg is either a 3d or a pointer to the tex obj w/ previous 3d passed.
  float *angles = (float *) mxGetData(prhs[1]);



  float *ts = (float *) mxGetData(prhs[2]);
  doFwdXform = (bool *) mxGetData(prhs[3]);

  if (*doFwdXform)
  {
    mexPrintf("Doing the forward xform\n");
    // transpose matrix
    shifts = make_float3(-ts[0],-ts[1],-ts[2]);
    
  }
  else
  {
    mexPrintf("Doing the inverse xform\n");
    shifts = make_float3(ts[0],ts[1],ts[2]);


  }

    rm_1   = make_float3(angles[0],angles[3],angles[6]);
    rm_2   = make_float3(angles[1],angles[4],angles[7]);
    rm_3   = make_float3(angles[2],angles[5],angles[8]);

//  mexPrintf("ts %f %f %f\n", shifts.x, shifts.y, shifts.z);
//mexPrintf("rm1 %f %f %f\n", rm_1.x, rm_1.y, rm_1.z);
//mexPrintf("rm2 %f %f %f\n", rm_2.x, rm_2.y, rm_2.z);
//mexPrintf("rm3 %f %f %f\n", rm_3.x, rm_3.y, rm_3.z);

  dims = make_int3(input_size[0],input_size[1],input_size[2]);
  



  // Create MX array and init with zeros
  outputArray = mxGPUCreateGPUArray(input_dims,
                                    input_size,
                                    mxSINGLE_CLASS,
                                    input_data_type,
                                    MX_GPU_INITIALIZE_VALUES);

  d_output_img = (float *)(mxGPUGetData(outputArray));


  if (make_texture_obj)
  {

    d_input_img = (float *)(mxGPUGetDataReadOnly(inputArray));

    hipArray *cuArray;
//    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    checkCudaErrors(hipMalloc3DArray(&cuArray,
                                      &channelDesc,
                                      make_hipExtent(dims.x,dims.y,dims.z)));

    hipMemcpy3DParms p = {0};
    p.extent = make_hipExtent(dims.x,dims.y,dims.z);
    p.srcPtr = make_hipPitchedPtr(d_input_img, dims.x*sizeof(float),dims.x,dims.y);
    p.dstArray = cuArray;
    p.kind = hipMemcpyDeviceToDevice;

    hipMemcpy3D(&p);

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;
   

//    size_t n_bytes = dims.x*dims.y*dims.z*sizeof(float);
//    float* buffer;
//    checkCudaErrors(hipMalloc(&buffer, n_bytes );
//    struct hipResourceDesc resDesc;

//    memset(&resDesc, 0, sizeof(hipResourceDesc));
//    resDesc.resType = hipResourceTypeLinear;
//    resDesc.res.linear.devPtr = buffer;
//    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
//    resDesc.res.linear.desc.x = 32; // bits per channel
//    resDesc.res.linear.sizeInBytes = n_bytes;

    struct hipTextureDesc texDesc;
    memset(&texDesc,0,sizeof(texDesc));

    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;

    
    mwSize const ptr_dims = 1;
    mwSize ptr_size[1];
    ptr_size[0] = (mwSize) 1;
    mxClassID output_data_class = {mxUINT64_CLASS};
    mxComplexity output_data_complexity = {mxREAL};

    // Forward and inverse transforms
    plhs[1] =  mxCreateNumericArray(ptr_dims,
                                    ptr_size,
                                    output_data_class,
                                    output_data_complexity);

    tex = (hipTextureObject_t *)mxGetData(plhs[1]);

    *tex = 0;

    // Make the arrays persistent
    mexMakeArrayPersistent(plhs[1]);
    // TODO does the cuArray need to be persistent?


    hipCreateTextureObject(tex,&resDesc,&texDesc,NULL);



  }



  dim3 dimBlock = dim3(16,16,1);
  dim3 dimGrid = dim3((dims.x+16-1) / 16, (dims.y+16-1)/16, dims.z);


//  mexPrintf("Dims: %d %d %d\nBlock: %d %d %d\nGrid: %d %d %d\n",
//            dims.x, dims.y, dims.z, dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

  float3 size_shift = make_float3(0.0f, 0.0f, 0.0f);
  if (IsEven(dims.x)) size_shift.x = 0.5f;
  if (IsEven(dims.y)) size_shift.y = 0.5f;
  if (IsEven(dims.z)) size_shift.z = 0.5f;

  transformKernel_FWD<< <dimGrid, dimBlock, 0, hipStreamPerThread>> >(*tex, d_output_img, dims,
                                              rm_1,rm_2,rm_3,shifts,EXTRAPVAL,*doFwdXform, size_shift);
  


  checkCudaErrors(hipDeviceSynchronize());


  // Check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");


  plhs[0] = mxGPUCreateMxArrayOnGPU(outputArray);

  mxGPUDestroyGPUArray(inputArray);
  mxGPUDestroyGPUArray(outputArray);

//  hipFreeArray(cuArray);
//  hipDestroyTextureObject(tex3d);

//  checkCudaErrors(hipFree(d_input_img));
//  checkCudaErrors(hipFree(d_output_img));


}
