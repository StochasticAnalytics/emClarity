#include "hip/hip_runtime.h"

#include "include/core_headers.cuh"



#define MAX_EPSILON_ERROR 5e-3f



////////////////////////////////////////////////////////////////////////////////
// Constants


// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;
const float EXTRAPVAL = 0.0f;
const int  slice_thickness_pixel_radius = 13; // TODO should

const float wanted_padding = 1.0; // oversample the 2d ctf

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void sf3dKernel(float *outputData,
                           uint3 dims,
                           float2 sinAcosA,
                           float extrapVal)
{

  // Assuming a single-Y-axis tilt such that the rotation is
  // [ c, 0, s,
  //   0, 1, 0,
  //  -s, 0, c]

  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  if (x >= dims.x) { return ; }
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
  if (y >= dims.y) { return ; }
  unsigned int z = blockIdx.z; //*blockDim.z + threadIdx.z;
  if (z >= dims.z) {  return ; }
  float u,w,tv,tu,tw,zWeight,frequency_squared;


  // First calc the Z-dimension, and check that we are close enough to the plane
  u = (float)x - (float)dims.x/2;
  w = (float)z - (float)dims.z/2;
  tw = -u*sinAcosA.x + w*sinAcosA.y;

  if (tw >= -slice_thickness_pixel_radius & tw <= slice_thickness_pixel_radius)
  {
    // FIXME this should approximate a sinc
    zWeight = 0.5 + 0.5*cosf(tw*PI/(float)(slice_thickness_pixel_radius+1.0f));

    tu =  u*sinAcosA.y + w*sinAcosA.x; 
    tu /= (float)dims.x; // Normalized coords
    tw /= (float)dims.z;
    
    // FIXME this extra division could go
//    u /= (float)(dims.x/2);
    //100.0f;

    tu += 0.5f;
    tw += 0.5f;
    tv = ((float)y - (float)dims.y/2) / (float)dims.y + 0.5f; 

    // TODO one of these is probably supposed to be inclusive
    if (tu > 0 & tw > 0 & tu > 0 & tu < 1 - 1/(float)dims.x & tw < 1 - 1/(float)dims.z)
    {
      // re-use u to calc a radial weight. Set u at origin to u(1) as in imod
      if (u == 0) { u = 0.2; }
      u /= (float)dims.x ;


      // TODO The radial weighting and exposure weighting can, and probably should just be done on the 2d ctf prior to texturing
      outputData[ (z*dims.y + y) * dims.x + x ] += ( zWeight * (fabsf(u)) * tex2D(tex, tu, tv));
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{


  // Initialize the MathWorks GPU API.
  mxInitGPU();

  // TODO if these aren't typed properly in the mex call, I can't cast them appropriatley here
  // TODO general angles, here assume single-Y-axis tilt
  /* Check for proper number of arguments. TODO add checks on narg and types*/
    if ( nrhs != 14 & nrhs != 15 ) 
    {
        mexErrMsgIdAndTxt("MATLAB:mexSF3D:rhs",
            "requires 14 inputs.");
    }

  bool * doHalfGrid =  (bool *) mxGetData(prhs[0]);  
  bool * doSqCTF    =  (bool *) mxGetData(prhs[1]); 
  uint * wantedSize =  (uint *) mxGetData(prhs[2]); // Should be a uint32 in matlab
  float* pixelSize  =  (float*) mxGetData(prhs[3]);
  float* waveLength =  (float*) mxGetData(prhs[4]);
  float* CS         =  (float*) mxGetData(prhs[5]);
  float* defocus1   =  (float*) mxGetData(prhs[6]);
  float* defocus2   =  (float*) mxGetData(prhs[7]);
  float* defocusAst =  (float*) mxGetData(prhs[8]);
  float* AmpContrast=  (float*) mxGetData(prhs[9]);
  uint*   nTilts     = (uint  *) mxGetData(prhs[10]);
  float* tiltAngle  =  (float*) mxGetData(prhs[11]);
  float* exposure   =  (float*) mxGetData(prhs[12]);
  float* occupancy  =  (float*) mxGetData(prhs[13]);
  int*   launch     = (int *) mxGetData(prhs[14]); // should be an int16 in matlab

  float * d_output_img = NULL;
  float * d_ctf_img = NULL;
  uint3 dims;
  uint2 ctf_dims;
  uint2 o_ctf_dims;

  float2 sinAcosA[*nTilts];


  ctfParams b_ctf(*doHalfGrid,*doSqCTF,*pixelSize,*waveLength,*CS,*AmpContrast,
                  *defocus1,  *defocus2, *defocusAst);

///* Check for proper number of arguments. TODO add checks on narg and types*/
//  if ( ! mxGPUIsValidGPUData(prhs[0]) ) 
//  {
//      mexErrMsgIdAndTxt("MATLAB:mexFFT:rhs",
//          "This inputArray is not valid gpudata.");
//  }


  mxGPUArray * outputArray;

  for (int iAng = 0; iAng < *nTilts; iAng++) 
  {

    sinAcosA[iAng] = make_float2(sinf(tiltAngle[iAng]*PI/180.0f),cosf(tiltAngle[iAng]*PI/180.0f));

  }
    

  dims     = make_uint3(wantedSize[0],wantedSize[1],wantedSize[2]);
  ctf_dims = make_uint2(wantedSize[0]*wanted_padding,wantedSize[1]*wanted_padding);

  // Calculate this prior to any half-dim reduction
  float2 fourierVoxelSize;
  fourierVoxelSize = make_float2( 1.0f/(*pixelSize * (float)ctf_dims.x), 
                                  1.0f/(*pixelSize * (float)ctf_dims.y));

  if (*doHalfGrid )
  {
    o_ctf_dims = make_uint2(0, ctf_dims.y/2);
    ctf_dims.x = ctf_dims.x/2 + 1;
  }
  else
  {
    o_ctf_dims = make_uint2(ctf_dims.x/2, ctf_dims.y/2);
  }



  long numel_output, numel_ctf;
  numel_output = dims.x * dims.y  *dims.z * sizeof(float);
  numel_ctf    = ctf_dims.x * ctf_dims.y * sizeof(float);

  // Allocate device memory for result
  mwSize output_dims = 3;
  mwSize output_size[3] = {dims.x, dims.y, dims.z};

  // Create MX array and init with zeros
  outputArray = mxGPUCreateGPUArray(output_dims,
                                    output_size,
                                    mxSINGLE_CLASS,
                                    mxREAL,
                                    MX_GPU_INITIALIZE_VALUES);

  d_output_img = (float *)(mxGPUGetData(outputArray));

  hipArray *cuArray;

  // TODO where does the 32 come from?
  hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);


//  mexit(__LINE__);
  checkCudaErrors(hipMallocArray(&cuArray,
                                  &channelDesc,
                                  ctf_dims.x,
                                  ctf_dims.y));

  // Set texture parameters
  // hipAddressModeWrap hipAddressModeClamp hipAddressModeMirror hipAddressModeBorder
  tex.addressMode[0] = hipAddressModeClamp;
  tex.addressMode[1] = hipAddressModeClamp;

  tex.filterMode = hipFilterModeLinear;

  tex.normalized = true;    // access with normalized texture coordinates

  // Params for the 2d ctf
  mwSize const   ctf_number_of_dims = 2;
  mxComplexity   ctf_type = mxREAL;
  mxClassID      ctf_class = mxSINGLE_CLASS;
  mwSize         ctf_size[2];
  ctf_size[0] = (mwSize)  ctf_dims.x;
  ctf_size[1] = (mwSize)  ctf_dims.y;
    
  mxGPUArray *ctfArray;  
  // TODO it would be nice not to init all the zeros, but then the fourier padding would need to be dealt with.
  ctfArray = mxGPUCreateGPUArray(ctf_number_of_dims,
                                 ctf_size,
                                 ctf_class,
                                 ctf_type,
                                 MX_GPU_INITIALIZE_VALUES);

  d_ctf_img = (hipfftReal *)(mxGPUGetData(outputArray));

  dim3 ctfBlock(32, 32, 1);
  dim3 ctfGrid(ctf_dims.x / ctfBlock.x, ctf_dims.y / ctfBlock.y, 1);
  bool calc_centered = true;
  bool radial_weight = false;
  for (int iAng = 0 ; iAng < *nTilts ; iAng ++)
  {


    // Create the 2d ctf
    ctf<<< ctfGrid, ctfBlock >>>(d_ctf_img, ctf_dims, o_ctf_dims, b_ctf, fourierVoxelSize,
                                 calc_centered, radial_weight, exposure[iAng]);


    // Put the ctf in tex2
    checkCudaErrors(hipMemcpyToArray(cuArray,
                                      0,
                                      0,
                                      d_ctf_img,
                                      numel_ctf,
                                      hipMemcpyDeviceToDevice));

    
    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

    // Call the sf3d kernel
    //TODO

  // Would specifying a 3d grid speed up by improving drop out over a block?
    int dimDist = 32;
    dim3 threads_per_block = dim3(dimDist,dimDist,1); // max is 1024 threads/block for 2.x --> 7.5 compute capability
    dim3 dimGrid = dim3((dims.x+dimDist-1) / dimDist,(dims.y+dimDist-1)/dimDist,dims.z);

    sf3dKernel<<<dimGrid, threads_per_block >>>(d_output_img, dims,
                                                  sinAcosA[iAng],EXTRAPVAL);
  }




  checkCudaErrors(hipDeviceSynchronize());


  // Check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");


  plhs[0] = mxGPUCreateMxArrayOnGPU(outputArray);


  mxGPUDestroyGPUArray(outputArray);


//  checkCudaErrors(hipFree(d_input_img));
//  checkCudaErrors(hipFree(d_output_img));


}
