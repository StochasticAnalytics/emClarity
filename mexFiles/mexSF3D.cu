#include "hip/hip_runtime.h"

#include "include/core_headers.cuh"



#define MAX_EPSILON_ERROR 5e-3f



////////////////////////////////////////////////////////////////////////////////
// Constants


// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;
const float  slice_thickness_pixel_radius = 4; // TODO should
const float cosine_edge_arg = PI / (float)slice_thickness_pixel_radius;
const float cosine_edge_norm = 1.0f / 4.0f;


////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void sf3dKernel(float *outputData,
                           float3 size_shift,
                           uint3 dims,
                           float2 sinAcosA
                           )
{

  // Assuming a single-Y-axis tilt such that the rotation is
  // [ c, 0, s,
  //   0, 1, 0,
  //  -s, 0, c]

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  if (x >= dims.x) { return ; }
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  if (y >= dims.y) { return ; }
//  int z = blockIdx.z*blockDim.z + threadIdx.z;
//  if (z >= dims.z) {  return ; }
  float u,w,tv,tu,tw,tw_pre,zWeight;


  // First calc the Z-dimension, and check that we are close enough to the plane
  u = (float)x - (float)dims.x/2;
  tw_pre = -u*sinAcosA.x + size_shift.z;
  tv = ((float)y - (float)(dims.y/2) + size_shift.y) / (float)dims.y + 0.5f;

  for (int z = 0; z < dims.z; z++)
  {
    w = (float)z - (float)(dims.z/2);
    tw = tw_pre + w*sinAcosA.y;

    if (tw < -slice_thickness_pixel_radius | tw > slice_thickness_pixel_radius)
    {
      continue;
    }

      // FIXME this should approximate a sinc
      zWeight = (0.5 + 0.5*cosf(tw * cosine_edge_arg )) * cosine_edge_norm;

      tu =  u*sinAcosA.y + w*sinAcosA.x + size_shift.x; 
      tu /= (float)dims.x; // Normalized coords
      tw /= (float)dims.z;
      
      // FIXME this extra division could go
  //    u /= (float)(dims.x/2);
      //100.0f;

      tu += 0.5f;
      tw += 0.5f;
       

      outputData[ (z*dims.y + y) * dims.x + x ] += ( zWeight * tex2D(tex, tu, tv));
  }

}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{


//  // Initialize the MathWorks GPU API.
//  mxInitGPU();

  // TODO if these aren't typed properly in the mex call, I can't cast them appropriatley here
  // TODO general angles, here assume single-Y-axis tilt
  /* Check for proper number of arguments. TODO add checks on narg and types*/
    if ( nrhs != 14 & nrhs != 15 ) 
    {
        mexErrMsgIdAndTxt("MATLAB:mexSF3D:rhs",
            "requires 14 inputs.");
    }

  bool * doHalfGrid =  (bool *) mxGetData(prhs[0]);  
  bool * doSqCTF    =  (bool *) mxGetData(prhs[1]); 
  uint * wantedSize =  (uint *) mxGetData(prhs[2]); // Should be a uint32 in matlab
  float* pixelSize  =  (float*) mxGetData(prhs[3]);
  float* waveLength =  (float*) mxGetData(prhs[4]);
  float* CS         =  (float*) mxGetData(prhs[5]);
  float* defocus1   =  (float*) mxGetData(prhs[6]);
  float* defocus2   =  (float*) mxGetData(prhs[7]);
  float* defocusAst =  (float*) mxGetData(prhs[8]);
  float* AmpContrast=  (float*) mxGetData(prhs[9]);
  uint*   nTilts     = (uint  *) mxGetData(prhs[10]);
  float* tiltAngle  =  (float*) mxGetData(prhs[11]);
  float* exposure   =  (float*) mxGetData(prhs[12]);
  float* occupancy  =  (float*) mxGetData(prhs[13]);
  int*   launch     = (int *) mxGetData(prhs[14]); // should be an int16 in matlab

  float * d_output_img = NULL;
  float * d_ctf_img = NULL;
  uint3 dims;
  uint2 ctf_dims;
  uint2 o_ctf_dims;

  float2 sinAcosA[*nTilts];



  mxGPUArray * outputArray;

  float sin_t = 0.0f;
  float cos_t = 0.0f;
  for (int iAng = 0; iAng < *nTilts; iAng++) 
  {

    sincosf( deg_2_rad(-tiltAngle[iAng]), &sin_t, &cos_t);
    sinAcosA[iAng] = make_float2(sin_t, cos_t);

  }
    

  dims     = make_uint3(wantedSize[0],wantedSize[1],wantedSize[2]);
  ctf_dims = make_uint2(wantedSize[0],wantedSize[1]);



  if (*doHalfGrid )
  {
    o_ctf_dims = make_uint2(0, ctf_dims.y/2);
    ctf_dims.x = ctf_dims.x/2 + 1;
  }
  else
  {
    o_ctf_dims = make_uint2(ctf_dims.x/2, ctf_dims.y/2);
  }




  // Allocate device memory for result
  mwSize output_dims = 3;
  mwSize output_size[3] = {dims.x, dims.y, dims.z};

  // Allocate device memory for the weights

  // Create MX array and init with zeros
  outputArray = mxGPUCreateGPUArray(output_dims,
                                    output_size,
                                    mxSINGLE_CLASS,
                                    mxREAL,
                                    MX_GPU_INITIALIZE_VALUES);

  d_output_img = (float *)(mxGPUGetData(outputArray));

  hipArray *cuArray;

  // TODO where does the 32 come from?
  hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);


//  mexit(__LINE__);
    // hipMallocArray( array_ptr, descriptor, width, height, flags.)
    // cuda arrays refer to memory speed width > height > depth (x > y > z)
  (hipMallocArray(&cuArray,
                   &channelDesc,
                   ctf_dims.x,
                   ctf_dims.y));

  // Set texture parameters
  // hipAddressModeWrap hipAddressModeClamp hipAddressModeMirror hipAddressModeBorder
  tex.addressMode[0] = hipAddressModeBorder;
  tex.addressMode[1] = hipAddressModeBorder;

  tex.filterMode = hipFilterModeLinear;

  tex.normalized = true;    // access with normalized texture coordinates

  // Params for the 2d ctf
  mwSize const   ctf_number_of_dims = 2;
  mxComplexity   ctf_type = mxREAL;
  mxClassID      ctf_class = mxSINGLE_CLASS;
  mwSize         ctf_size[2];
  ctf_size[0] = (mwSize)  ctf_dims.x;
  ctf_size[1] = (mwSize)  ctf_dims.y;
    
  mxGPUArray *ctfArray;  
  // TODO it would be nice not to init all the zeros, but then the fourier padding would need to be dealt with.
  ctfArray = mxGPUCreateGPUArray(ctf_number_of_dims,
                                 ctf_size,
                                 ctf_class,
                                 ctf_type,
                                 MX_GPU_INITIALIZE_VALUES);

  d_ctf_img = (float *)(mxGPUGetData(ctfArray));




  // Would specifying a 3d grid speed up by improving drop out over a block?
  
    dim3 threads_per_block = dim3(32,32,1); // max is 1024 threads/block for 2.x --> 7.5 compute capability
    dim3 dimGrid = dim3((dims.x + threads_per_block.x -1) / threads_per_block.x,
                        (dims.y + threads_per_block.y -1) / threads_per_block.y,
                        1);//(dims.z + threads_per_block.z -1) / threads_per_block.z);

    dim3 ctfBlock(32, 32, 1);
    dim3 ctfGrid((ctf_dims.x + ctfBlock.x - 1) / ctfBlock.x, (ctf_dims.y + ctfBlock.y - 1) / ctfBlock.y, 1);
    bool calc_centered = true;

  float3 size_shift = make_float3(0.0f, 0.0f, 0.0f);
  if (IsEven(dims.x)) size_shift.x = 0.5f;
  if (IsEven(dims.y)) size_shift.y = 0.5f;
  if (IsEven(dims.z)) size_shift.z = 0.5f;

    // Calculate this prior to any half-dim reduction
    float2 fourierVoxelSize;
    fourierVoxelSize = make_float2( 1.0f/(pixelSize[0] * (float)ctf_dims.x), 
                                    1.0f/(pixelSize[0] * (float)ctf_dims.y));


  for (int iAng = 0 ; iAng < *nTilts ; iAng ++)
  {

    if (iAng > 0)
    {
      fourierVoxelSize.x = 1.0f/(pixelSize[iAng] * (float)ctf_dims.x);
      fourierVoxelSize.y = 1.0f/(pixelSize[iAng] * (float)ctf_dims.y);
    }



    ctfParams b_ctf(*doHalfGrid,*doSqCTF,pixelSize[iAng],waveLength[iAng],CS[iAng],AmpContrast[iAng],
                    defocus1[iAng],  defocus2[iAng], defocusAst[iAng]);


    // Create the 2d ctf
    ctf<<< ctfGrid, ctfBlock ,0,hipStreamPerThread >>>(d_ctf_img, ctf_dims, o_ctf_dims, b_ctf, fourierVoxelSize,
                                 calc_centered, occupancy[iAng], exposure[iAng]);


    // Put the ctf in tex2

    (hipMemcpy2DToArray (cuArray, // Destination ptr
                          0, // Destination starting X offset (columns in bytes) 
                          0, // Destination starting Y offset (rows) 
                          d_ctf_img, // src ptr
                          ctf_dims.x * sizeof(float), // Pitch of source memory
                          ctf_dims.x * sizeof(float), // Width of matrix transfer (columns in bytes)  
                          ctf_dims.y, // Height of matrix transfer (rows) 
                          hipMemcpyDeviceToDevice));

    
    // Bind the array to the texture FIXME this is deprecated, switch to a texture object ... I think I use this in interpolation mexXform3d
    (hipBindTextureToArray(tex, cuArray, channelDesc));

    // Call the sf3d kernel
    sf3dKernel<<<dimGrid, threads_per_block,0,hipStreamPerThread >>>(d_output_img, size_shift,
                                                dims, sinAcosA[iAng]);



    // FIXME if you could bin an array of texture objects, you could launch the sf3dKernel outside the loop once. 
  }

  (hipStreamSynchronize(hipStreamPerThread));

//    (hipUnbindTexture(tex));




  // Check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");


  plhs[0] = mxGPUCreateMxArrayOnGPU(outputArray);


  (hipFreeArray(cuArray));
  mxGPUDestroyGPUArray(outputArray);

  mxGPUDestroyGPUArray(ctfArray);


//  (hipFree(d_input_img));
//  (hipFree(d_output_img));


}
