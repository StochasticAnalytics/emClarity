#include "hip/hip_runtime.h"

#include "include/core_headers.cuh"

// #define mexFP16_DEBUG_PRINT(args) mexPrintf("%s\n", args)
#define mexFP16_DEBUG_PRINT(...)

__global__ void convert_fp16_to_fp32(const uint16_t* __restrict__ input_half, float* __restrict__ output_single, const int N) {
  // Could be improved with a simple vector load.
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x;  idx < N; idx += gridDim.x * blockDim.x) 
    output_single[idx] = __half2float(__ushort_as_half(input_half[idx]));
  
}

__global__ void convert_fp32_to_fp16(const float* __restrict__ input_single, uint16_t* __restrict__ output_half, const int N) {
  // Could be improved with a simple vector store.
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x;  idx < N; idx += gridDim.x * blockDim.x) 
     output_half[idx] = __half_as_ushort(__float2half_rn(input_single[idx]));
  
}



void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]) {

  if (nrhs != 4) {
    mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                      "This function requires 2 input matrices, a boolean (to_half), and an int (*n_elements).");
  }
  bool single_array_is_on_gpu = false;
  bool half_array_is_on_gpu = false;
  // First check to see if we have a gpu arra
  if (mxIsGPUArray(prhs[0])) {
    // Now let's see if it isvalid data
    if (!mxGPUIsValidGPUData(prhs[0])) {
      mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                        "This function requires the first input to be a valid gpuArray.");
    }
    // And if it is single precision ( written this way causes a segfault, trust that emc_halfcast has done the checking.)
    // if (mxGPUGetClassID((const mxGPUArray *)prhs[0]) != mxSINGLE_CLASS) {
    //   mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
    //                     "This function requires the first input to be of type single mxGPUGetClassID.");
    // }
    single_array_is_on_gpu = true;  
  }
  
  // Same thing for the half array
  if (mxIsGPUArray(prhs[1])) {
    if (!mxGPUIsValidGPUData(prhs[1])) {
      mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                        "This function requires the second input to be a valid gpuArray.");
    }
    //( written this way causes a segfault, trust that emc_halfcast has done the checking.)
    // if (mxGPUGetClassID((const mxGPUArray *)prhs[1]) != mxUINT16_CLASS) {
    //    mexPrintf("Here %d\n", __LINE__);
    //   mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
    //                     "This function requires the second input to be of type uint16 mxGPUGetClassID.");
    // }
    half_array_is_on_gpu = true;
  }

  if (!mxIsLogical(prhs[2])) {
    mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                      "This function requires the third input to be of type logical.");
  }

  if (!mxIsInt64(prhs[3])) {
    mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                      "This function requires the fourth input to be of type int.");
  }


  float* input_single;
  uint16_t* input_uint16;

  // mxGPUCreateFromMxArray will return a read only pointer if the underlying is a matlab gpuArray
  // To avoid a copy but still get a pointer we can cast the const away.
  // This seems dodgy as fuck
  if (single_array_is_on_gpu) {
    mxGPUArray const * inputArray  = mxGPUCreateFromMxArray(prhs[0]);
    input_single =  (float *) mxGPUGetData((mxGPUArray *)inputArray);
  }
  else
    input_single =  (float *) mxGetData(prhs[0]);

  if (half_array_is_on_gpu) {
    mxGPUArray const * inputArray  = mxGPUCreateFromMxArray(prhs[1]);
    input_uint16 = (uint16_t *) mxGPUGetData((mxGPUArray *)inputArray);
  }
  else
    input_uint16 = (uint16_t *) mxGetData(prhs[1]);

  bool* cast_to_fp16 = (bool *) mxGetData(prhs[2]);
  size_t* n_elements = (size_t *) mxGetData(prhs[3]);



  // First, we need to take care of any data conversion needed.
  float* temporary_single;
  uint16_t* temporary_uint16;

  // Output half on gpu so we need to use the conversion kernel
  if (cast_to_fp16[0]) {
    mexFP16_DEBUG_PRINT("Casting to half\n");
    if (half_array_is_on_gpu) {
      mexFP16_DEBUG_PRINT("Casting to half that is on device already\n");
      temporary_uint16 = input_uint16;
      if (single_array_is_on_gpu) {
        temporary_single = input_single;
      } 
      else {
        mexFP16_DEBUG_PRINT("Copying single to device\n");
        checkCudaErrors(hipMallocAsync(&temporary_single, *n_elements * sizeof(float), hipStreamPerThread));
        checkCudaErrors(hipMemcpyAsync(temporary_single, input_single, *n_elements  * sizeof(float), hipMemcpyHostToDevice, hipStreamPerThread));
      }
      convert_fp32_to_fp16<<<1, 1, 0, hipStreamPerThread>>>(temporary_single, temporary_uint16, *n_elements);

      if (!single_array_is_on_gpu)
        checkCudaErrors(hipFreeAsync(temporary_single, hipStreamPerThread));

      checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
    }
    else {
      mexFP16_DEBUG_PRINT("Casting to half that is on host already\n");
      // Even though we could do this in place, then only copy over the half precision data to the host,
      // we want to keep the input data un altered.
      if (single_array_is_on_gpu) {
        mexFP16_DEBUG_PRINT("Copying single to host\n");
        checkCudaErrors(hipHostMalloc(&temporary_single, *n_elements * sizeof(float)));
        checkCudaErrors(hipMemcpyAsync(temporary_single, input_single, *n_elements  * sizeof(float), hipMemcpyDeviceToHost, hipStreamPerThread));
        checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
      } 
      else 
        temporary_single = input_single;

      mexFP16_DEBUG_PRINT(std::to_string(*n_elements));
      half_float::half* half_ptr = (half_float::half *)(input_uint16);

      for (int i = 0; i < *n_elements ; i++) {
        half_ptr[i] = half_float::half(temporary_single[i]);
      }

      if (single_array_is_on_gpu)
        checkCudaErrors(hipHostFree(temporary_single));

    }
  }
  else {
    mexFP16_DEBUG_PRINT("Casting to single\n");
    // Casting from half to single
    if (single_array_is_on_gpu) {
      temporary_single = input_single;
      if (half_array_is_on_gpu) {
        temporary_uint16 = input_uint16;
      } 
      else {
        checkCudaErrors(hipMallocAsync(&temporary_uint16, *n_elements * sizeof(uint16_t), hipStreamPerThread));
        checkCudaErrors(hipMemcpyAsync(temporary_uint16, input_uint16, *n_elements  * sizeof(uint16_t), hipMemcpyHostToDevice, hipStreamPerThread));
      }
      convert_fp16_to_fp32<<<1024, 32, 0, hipStreamPerThread>>>(temporary_uint16, temporary_single, *n_elements);

      if (!half_array_is_on_gpu)
        checkCudaErrors(hipFreeAsync(temporary_uint16, hipStreamPerThread));

     checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
    }
    else {
      mexFP16_DEBUG_PRINT("Casting to single that is on host already\n");
      // the output single is on the host
      half_float::half* half_ptr = (half_float::half *)(input_uint16);
      if (half_array_is_on_gpu) {
        mexFP16_DEBUG_PRINT("Copying half to host\n");
        checkCudaErrors(hipHostMalloc(&temporary_uint16, *n_elements * sizeof(uint16_t)));
        checkCudaErrors(hipMemcpyAsync(temporary_uint16, input_uint16, *n_elements  * sizeof(uint16_t), hipMemcpyDeviceToHost, hipStreamPerThread));
        checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
        half_ptr = (half_float::half *)(temporary_uint16);
      } 

      for (int i = 0; i < *n_elements ; i++) {
        float tmp = half_float::half_cast<float>(half_ptr[i]);
        input_single[i] = half_float::half_cast<float>(half_ptr[i]);
      }

      if (half_array_is_on_gpu)
        checkCudaErrors(hipHostFree(temporary_uint16));
    }
  } // end if if cast to fp16 else to single


  return;

    
}