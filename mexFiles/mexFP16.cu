#include "hip/hip_runtime.h"

#include "include/core_headers.cuh"
#include <memory>

// #define mexFP16_DEBUG_PRINT(args) mexPrintf("%s\n", args)
#define mexFP16_DEBUG_PRINT(...)

__global__ void convert_fp16_to_fp32(const uint16_t* __restrict__ input_half, float* __restrict__ output_single, const int N) {
  // Could be improved with a simple vector load.
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x;  idx < N; idx += gridDim.x * blockDim.x) 
    output_single[idx] = __half2float(__ushort_as_half(input_half[idx]));
  
}

__global__ void convert_fp32_to_fp16(const float* __restrict__ input_single, uint16_t* __restrict__ output_half, const int N) {
  // Could be improved with a simple vector store.
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x;  idx < N; idx += gridDim.x * blockDim.x) 
     output_half[idx] = __half_as_ushort(__float2half_rn(input_single[idx]));
  
}



void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]) {

  if (nrhs != 4) {
    mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                      "This function requires 2 input matrices, a boolean (to_half), and an int (*n_elements).");
  }
  bool single_array_is_on_gpu = false;
  bool half_array_is_on_gpu = false;
  // First check to see if we have a gpu arra
  if (mxIsGPUArray(prhs[0])) {
    // Now let's see if it isvalid data
    if (!mxGPUIsValidGPUData(prhs[0])) {
      mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                        "This function requires the first input to be a valid gpuArray.");
    }
    // And if it is single precision ( written this way causes a segfault, trust that emc_halfcast has done the checking.)
    // if (mxGPUGetClassID((const mxGPUArray *)prhs[0]) != mxSINGLE_CLASS) {
    //   mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
    //                     "This function requires the first input to be of type single mxGPUGetClassID.");
    // }
    single_array_is_on_gpu = true;  
  }
  
  // Same thing for the half array
  if (mxIsGPUArray(prhs[1])) {
    if (!mxGPUIsValidGPUData(prhs[1])) {
      mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                        "This function requires the second input to be a valid gpuArray.");
    }
    //( written this way causes a segfault, trust that emc_halfcast has done the checking.)
    // if (mxGPUGetClassID((const mxGPUArray *)prhs[1]) != mxUINT16_CLASS) {
    //    mexPrintf("Here %d\n", __LINE__);
    //   mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
    //                     "This function requires the second input to be of type uint16 mxGPUGetClassID.");
    // }
    half_array_is_on_gpu = true;
  }

  if (!mxIsLogical(prhs[2])) {
    mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                      "This function requires the third input to be of type logical.");
  }

  if (!mxIsInt64(prhs[3])) {
    mexErrMsgIdAndTxt("MATLAB:mexFP16:rhs",
                      "This function requires the fourth input to be of type int.");
  }


  float* input_single;
  uint16_t* input_uint16;

  // mxGPUCreateFromMxArray will return a read only pointer if the underlying is a matlab gpuArray
  // To avoid a copy but still get a pointer we can cast the const away.
  // This seems dodgy as fuck
  if (single_array_is_on_gpu) {
    // mexEvalString("pause(3)");
    mxGPUArray const * inputArray  = mxGPUCreateFromMxArray(prhs[0]);
    input_single =  (float *) mxGPUGetData((mxGPUArray *)inputArray);
    // if we don't destroy this array we have a bad memory leak, yet it also doesn't seem like more memory is allocated
    // inside this block based on pausing and watching.
    mxGPUDestroyGPUArray(inputArray);
  }
  else
    input_single =  (float *) mxGetData(prhs[0]);

  if (half_array_is_on_gpu) {
    mxGPUArray const * inputArray  = mxGPUCreateFromMxArray(prhs[1]);
    input_uint16 = (uint16_t *) mxGPUGetData((mxGPUArray *)inputArray);
    mxGPUDestroyGPUArray(inputArray);
  }
  else
    input_uint16 = (uint16_t *) mxGetData(prhs[1]);

  bool* cast_to_fp16 = (bool *) mxGetData(prhs[2]);
  size_t* n_elements = (size_t *) mxGetData(prhs[3]);

  const size_t threads = 1024;
  const size_t blocks = (threads / *n_elements + 1024 - 1) /threads;


  // First, we need to take care of any data conversion needed.
  float* temporary_single;
  uint16_t* temporary_uint16;

  // Output half on gpu so we need to use the conversion kernel
  if (cast_to_fp16[0]) {
    mexFP16_DEBUG_PRINT("Casting to half\n");
    if (half_array_is_on_gpu) {
      mexFP16_DEBUG_PRINT("Casting to half that is on device already\n");
      temporary_uint16 = input_uint16;
      if (single_array_is_on_gpu) {
        temporary_single = input_single;
      } 
      else {
        mexFP16_DEBUG_PRINT("Copying single to device\n");
        checkCudaErrors(hipMallocAsync(&temporary_single, *n_elements * sizeof(float), hipStreamPerThread));
        checkCudaErrors(hipMemcpyAsync(temporary_single, input_single, *n_elements  * sizeof(float), hipMemcpyHostToDevice, hipStreamPerThread));
      }
      
      convert_fp32_to_fp16<<<threads, blocks, 0, hipStreamPerThread>>>(temporary_single, temporary_uint16, *n_elements);

      if (!single_array_is_on_gpu)
        checkCudaErrors(hipFreeAsync(temporary_single, hipStreamPerThread));

      checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
    }
    else {
      mexFP16_DEBUG_PRINT("Casting to half that is on host already\n");
      // Even though we could do this in place, then only copy over the half precision data to the host,
      // we want to keep the input data un altered.
      if (single_array_is_on_gpu) {
        mexFP16_DEBUG_PRINT("Copying single to host\n");
        checkCudaErrors(hipHostMalloc(&temporary_single, *n_elements * sizeof(float)));
        checkCudaErrors(hipMemcpy(temporary_single, input_single, *n_elements  * sizeof(float), hipMemcpyDeviceToHost));
        checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
      } 
      else 
        temporary_single = input_single;

      mexFP16_DEBUG_PRINT(std::to_string(*n_elements));
      half_float::half* half_ptr = (half_float::half *)(input_uint16);

      for (int i = 0; i < *n_elements ; i++) {
        half_ptr[i] = half_float::half(temporary_single[i]);
      }

      if (single_array_is_on_gpu)
        checkCudaErrors(hipHostFree(temporary_single));

    }
  }
  else {
    mexFP16_DEBUG_PRINT("Casting to single\n");
    // Casting from half to single
    if (single_array_is_on_gpu) {
      temporary_single = input_single;
      if (half_array_is_on_gpu) {
        temporary_uint16 = input_uint16;
      } 
      else {
        checkCudaErrors(hipMallocAsync(&temporary_uint16, *n_elements * sizeof(uint16_t), hipStreamPerThread));
        checkCudaErrors(hipMemcpyAsync(temporary_uint16, input_uint16, *n_elements  * sizeof(uint16_t), hipMemcpyHostToDevice, hipStreamPerThread));
      }
      convert_fp16_to_fp32<<<threads, blocks, 0, hipStreamPerThread>>>(temporary_uint16, temporary_single, *n_elements);

      if (!half_array_is_on_gpu)
        checkCudaErrors(hipFreeAsync(temporary_uint16, hipStreamPerThread));

     checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
    }
    else {
      mexFP16_DEBUG_PRINT("Casting to single that is on host already\n");
      // the output single is on the host
      half_float::half* half_ptr = (half_float::half *)(input_uint16);
      if (half_array_is_on_gpu) {
        mexFP16_DEBUG_PRINT("Copying half to host\n");
        checkCudaErrors(hipHostMalloc(&temporary_uint16, *n_elements * sizeof(uint16_t)));
        checkCudaErrors(hipMemcpy(temporary_uint16, input_uint16, *n_elements  * sizeof(uint16_t), hipMemcpyDeviceToHost));
        checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
        half_ptr = (half_float::half *)(temporary_uint16);
      } 

      for (int i = 0; i < *n_elements ; i++) {
        float tmp = half_float::half_cast<float>(half_ptr[i]);
        input_single[i] = half_float::half_cast<float>(half_ptr[i]);
      }

      if (half_array_is_on_gpu)
        checkCudaErrors(hipHostFree(temporary_uint16));
    }
  } // end if if cast to fp16 else to single


  return;

    
}