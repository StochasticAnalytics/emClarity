#include "hip/hip_runtime.h"
//% Matlab side code

#include "include/core_headers.cuh"


static __global__ void RealScale(hipfftReal*, float ) ;

//static void cleanUpMemory(void);
//{
//      mexPrintf("Destroying the plans\n");
//      hipfftDestroy(*plan);
//      hipfftDestroy(*planInv);
//      mxGPUDestroyGPUArray(inputArray); 
//}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{



///* Check for proper number of arguments. */
//  if ( nrhs != 2) {
//      mexErrMsgIdAndTxt("MATLAB:matrixDivide:rhs",
//          "This function requires 2 input matrices.");
//  }

  // Pointers to pass to cufft
  hipfftReal *pReal;
  hipfftComplex *pComplex;
  mxGPUArray *outputArray;  
  int *invTrim;
  // Handle for the plan, and check to see if it exists
  hipfftHandle *plan;
  hipfftHandle *planInv;
  bool make_plan = true;
  bool fwd_xform = true; 
  bool do_scale = false; // Currently just as fast to do it in matlab.

/* Check for proper number of arguments. */
  if ( ! mxGPUIsValidGPUData(prhs[0]) ) 
  {

      mexErrMsgIdAndTxt("MATLAB:mexFFT:rhs",
          "This inputArray is not valid gpudata.");
  }

  // Input array, could also just pass the dimensions
  mxGPUArray const * inputArray  = mxGPUCreateFromMxArray(prhs[0]);

//  mxGPUArray const * mex_EO = mxGPUCreateFromMxArray(prhs[1]);
  invTrim =  (int *) mxGetData(prhs[1]);

  // Get the size to use in making the plan
  mwSize const   input_dims = mxGPUGetNumberOfDimensions(inputArray);
  mwSize const * input_size = mxGPUGetDimensions(inputArray);
  mwSize const   numel_input = mxGPUGetNumberOfElements(inputArray);
  mxComplexity   input_type = mxGPUGetComplexity(inputArray);

  // Assuming we are only doing either R2C or C2R
  mxComplexity output_data_type;
  if ( input_type == mxREAL ) {
//    mexPrintf("It is real bitches\n");
    output_data_type =  mxCOMPLEX ;
    pReal = (hipfftReal *)(mxGPUGetDataReadOnly(inputArray));
  }
  else if (input_type == mxCOMPLEX) {
    fwd_xform = false;

    output_data_type = mxREAL ;
    pComplex = (hipfftComplex *)(mxGPUGetDataReadOnly(inputArray));
  }
  else
  {
    mexErrMsgIdAndTxt("MATLAB:mexFFT:rhs",
          "This inputArray is not real or complex.");
  }

  if (nrhs > 2)
  {
    make_plan = false;


    plan    = (hipfftHandle*) mxGetData(prhs[2]); 
    planInv = (hipfftHandle*) mxGetData(prhs[3]); 

    if (numel_input == 1)
    {
      size_t dummy;
//      mexPrintf("Destroying the plans\n");
      if (hipfftGetSize(*plan, &dummy) != HIPFFT_INVALID_PLAN)
        hipfftDestroy(*plan);
      // else
      //   mexPrintf("The fwd plan is not valid in destructor\n");
      if (hipfftGetSize(*planInv, &dummy) != HIPFFT_INVALID_PLAN)
        hipfftDestroy(*planInv);
      // else
      //   mexPrintf("The inv plan is not valid in destructor\n");
      if (mxGPUIsValidGPUData(prhs[0]) )
        mxGPUDestroyGPUArray(inputArray);
      // else
      //   mexPrintf("The inputArray is not valid in destructor\n");
      return;
    }
  } 
  
  // This is only used in the inverse xform. Probably a better way. is this safe?
  // This also should probably be calculated just once in the fftTransformer class and passed in.
 
  // All transforms are done out of place, so no FFTW pafft_dimsing is assumed.
  mwSize  output_size[input_dims];
  if (input_dims > 2)
  {
    if (input_size[2] > 1)
    {
      // 3d xform 
      if (fwd_xform) { output_size[0] = input_size[0]/2+1; }
      else { output_size[0] = input_size[0]*2 - *invTrim; } 
      output_size[1] = input_size[1];
      output_size[2] = input_size[2];   
    }
    else
    {
    // 2d xform
      if (fwd_xform) { output_size[0] = input_size[0]/2+1; }
      else { output_size[0] = input_size[0]*2 - *invTrim; }
      output_size[1] = input_size[1];
      output_size[2] = 1;
    }
  }
  else if (input_dims > 1)
  {
    if (input_size[1] > 1)
      {
      // also 2d
      if (fwd_xform) { output_size[0] = input_size[0]/2+1; }
      else { output_size[0] = input_size[0]*2 - *invTrim; }
      output_size[1] = input_size[1];     
      }
    else
    {
      if (fwd_xform) { output_size[0] = input_size[0]/2+1; }
      else { output_size[0] = input_size[0]*2 - *invTrim; }
      output_size[1] = 1;   
    }
  }




  // int xFormRank;
  int fft_dims[input_dims];
  // int batchSize;
  
  if (input_dims > 2) 
  { fft_dims[2] = (int) input_size[0];
    fft_dims[1] = (int) input_size[1];
    fft_dims[0] = (int) input_size[2];
  }
  else
  {
    fft_dims[1] = (int) input_size[0];
    fft_dims[0] = (int) input_size[1];
  }

  // if (input_dims > 2) 
  // {
  //   if (fft_dims[2] > 1) { xFormRank = 3; batchSize = fft_dims[2]; } else { xFormRank = 2; batchSize = fft_dims[1]; }
  // }
  // else
  // {
  //   if (fft_dims[1] > 1) { xFormRank = 2; batchSize = fft_dims[1]; } else { xFormRank = 1; batchSize = 1;}
  // }

  outputArray = mxGPUCreateGPUArray(input_dims,
                              output_size,
                              mxGPUGetClassID(inputArray),
                              output_data_type,
                              MX_GPU_DO_NOT_INITIALIZE);

  if (fwd_xform)
  {
    pComplex = (hipfftComplex *)(mxGPUGetData(outputArray));
  }
  else
  {
    pReal = (hipfftReal *)(mxGPUGetData(outputArray));
  }


  mwSize const numel_output = mxGPUGetNumberOfElements(outputArray);



  if (make_plan)
  {

    mwSize const ptr_dims = 1;
    mwSize ptr_size[1];
    ptr_size[0] = (mwSize) 1;
    mxClassID output_data_class = {mxUINT32_CLASS};
    mxComplexity output_data_complexity = {mxREAL};

    // Forward and inverse transforms
    plhs[1] =  mxCreateNumericArray(ptr_dims,
                                    ptr_size,
                                    output_data_class,
                                    output_data_complexity);
    plhs[2] =  mxCreateNumericArray(ptr_dims,
                                    ptr_size,
                                    output_data_class,
                                    output_data_complexity);

    plan    = (hipfftHandle*)mxGetData(plhs[1]);
    planInv = (hipfftHandle*)mxGetData(plhs[2]);

    // Make the arrays persistent
    mexMakeArrayPersistent(plhs[1]);
    mexMakeArrayPersistent(plhs[2]);


    if ( ! fwd_xform ) {
      mexErrMsgIdAndTxt("The cufft plan must be created on a forward transform.","womp");
    }

    if (input_dims == 3) {
       hipfftPlan3d(plan, fft_dims[0], fft_dims[1], fft_dims[2],  HIPFFT_R2C);
       hipfftPlan3d(planInv, fft_dims[0], fft_dims[1],fft_dims[2], HIPFFT_C2R);
    }
    else if (input_dims == 2) {
       hipfftPlan2d(plan, fft_dims[0], fft_dims[1],  HIPFFT_R2C);
       hipfftPlan2d(planInv, fft_dims[0], fft_dims[1], HIPFFT_C2R);
    }
    else {
      mexErrMsgIdAndTxt("The cufft plan must be created on a 1, 2, or 3 dimensional array.","womp");
    }


    // hipfftPlanMany(plan,    xFormRank, fft_dims, 
    //               NULL, NULL, NULL, NULL, NULL, NULL,
    //               HIPFFT_R2C, 1);
    // hipfftPlanMany(planInv, xFormRank, fft_dims, 
    //               NULL, NULL, NULL, NULL, NULL, NULL,
    //               HIPFFT_C2R, 1);

  }

  // Do the fft
  if (fwd_xform)
  {
//    mexPrintf("Doing the forward xform\n");
    hipfftExecR2C(*plan, (hipfftReal *)pReal, (hipfftComplex *)pComplex);
  }
  else
  {
//    mexPrintf("Doing the inverse xform\n");
    hipfftExecC2R(*planInv, pComplex, pReal);
    // Should probably be an option, but enforce scaling on the inverse
    // I don't know how to properly chose the block/thread configuration.
    if (do_scale)
    {
      RealScale<<<512,256>>>(pReal, numel_output);
    }
  }

  // mxGPUCreateMxArrayOnGPU is used to wrap the output data in an MxArray that can be passed back to matlab.
  plhs[0] = mxGPUCreateMxArrayOnGPU(outputArray);


//  mexPrintf("Afft_dimsress of plan is %d\n", *plan);
//  mexPrintf("Afft_dimsress of planInv is %d\n", *planInv);
  mxGPUDestroyGPUArray(inputArray);
  mxGPUDestroyGPUArray(outputArray);
//  mxGPUDestroyGPUArray(mex_EO);
}

// Complex scale
static __global__ void RealScale(hipfftReal *a,  float n_elements) 
{



  // Other dimensions?   
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  const float scaleBy = 1/n_elements;

  for (int i = threadID; i < n_elements; i += numThreads) {
    a[i] *= scaleBy;
  }



}


